#include <fstream>
#include <iostream>
#include <sstream>

#include "GpuTimer.cuh"
#include "VectorOperations.cuh"

#define MAX_ITER 1000
#define EPS 1e-4

namespace patch {
template <typename T>
std::string to_string(const T &n) {
    std::ostringstream stm;
    stm << n;
    return stm.str();
}
}  // namespace patch

void solveCG(float *A, float *b, float *x, float *p, float *r, float *tmp, float *tmp_scal, float *alpha, float *beta, float *r_norm, float *r_norm_old, float *h_r_norm) {
    dim3 mat_grid_dim((SIZE + BLOCK_ELEM_MAT - 1) / BLOCK_ELEM_MAT, (SIZE + BLOCK_SIZE_MAT - 1) / BLOCK_SIZE_MAT);

    vecDotVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(r, r, r_norm_old);
    int it = 0;
    while ((it < MAX_ITER) && (*h_r_norm > EPS)) {
        // Get Ap (tmp)
        matDotVec<<<mat_grid_dim, BLOCK_SIZE_MAT>>>(A, p, tmp);

        // Get alpha_k
        vecDotVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(p, tmp, tmp_scal);
        div<<<1, 1>>>(r_norm_old, tmp_scal, alpha);

        // Get r_{k + 1}
        scalDotVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(alpha, tmp, tmp);
        vecMinVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(r, tmp, r);

        // Get x_{k + 1}
        scalDotVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(alpha, p, tmp);
        vecPlusVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(x, tmp, x);

        // r_{k + 1} is small??

        // Get beta_{k}
        vecDotVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(r, r, r_norm);
        div<<<1, 1>>>(r_norm, r_norm_old, beta);

        // Get p_{k + 1}
        scalDotVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(beta, p, tmp);
        vecPlusVec<<<VEC_GRID_DIM, BLOCK_DIM_VEC>>>(r, tmp, p);

        vecCpy<<<1, 1>>>(r_norm, r_norm_old);

        hipMemcpy(h_r_norm, r_norm, sizeof(float), hipMemcpyDeviceToHost);
        it++;
    }
}

int main() {
    float *A = (float *)malloc(SIZE * SIZE * sizeof(float));
    float *b = (float *)malloc(SIZE * sizeof(float));
    float *x = (float *)malloc(SIZE * sizeof(float));
    float *h_r_norm = (float *)malloc(sizeof(float));
    *h_r_norm = 1.0;

    fillA(A);
    fillb(b);

    float *dev_A, *dev_b, *dev_x, *dev_p, *dev_r, *dev_tmp;
    hipMalloc((void **)&dev_A, SIZE * SIZE * sizeof(float));
    hipMalloc((void **)&dev_b, SIZE * sizeof(float));
    hipMalloc((void **)&dev_x, SIZE * sizeof(float));
    hipMalloc((void **)&dev_p, SIZE * sizeof(float));
    hipMalloc((void **)&dev_r, SIZE * sizeof(float));
    hipMalloc((void **)&dev_tmp, SIZE * sizeof(float));

    hipMemcpy(dev_A, A, SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x, SIZE * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dev_p, b, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_r, b, SIZE * sizeof(float), hipMemcpyHostToDevice);

    float *dev_alpha, *dev_beta, *dev_r_norm, *dev_r_norm_old, *dev_tmp_scal;
    hipMalloc((void **)&dev_alpha, sizeof(float));
    hipMalloc((void **)&dev_beta, sizeof(float));
    hipMalloc((void **)&dev_r_norm, sizeof(float));
    hipMalloc((void **)&dev_r_norm_old, sizeof(float));
    hipMalloc((void **)&dev_tmp_scal, sizeof(float));

    GpuTimer timing;

    timing.Start();
    solveCG(dev_A, dev_b, dev_x, dev_p, dev_r, dev_tmp, dev_tmp_scal, dev_alpha, dev_beta, dev_r_norm, dev_r_norm_old, h_r_norm);

    hipDeviceSynchronize();
    timing.Stop();

    double res = timing.GetTime();

    std::cout << "Elapsed time: " << res << std::endl;

    std::ofstream outfile("Performance/Threads_" + patch::to_string(BLOCK_DIM_VEC) + "/" + patch::to_string(SIZE));

    outfile << timing.GetTime() << std::endl;

    outfile.close();

    hipMemcpy(x, dev_x, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // print1DVec(x);

    free(A);
    free(b);
    free(x);
    free(h_r_norm);

    hipFree(dev_A);
    hipFree(dev_b);
    hipFree(dev_x);
    hipFree(dev_p);
    hipFree(dev_r);
    hipFree(dev_tmp);

    hipFree(dev_alpha);
    hipFree(dev_beta);
    hipFree(dev_r_norm);
    hipFree(dev_r_norm_old);
    hipFree(dev_tmp_scal);
}